#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <hiprand/hiprand_kernel.h>
#include "optix_estimator.h"
#include <optix.h>
#include <optix_stubs.h>
#include <optix_stack_size.h>
#include <optix_function_table_definition.h>
#include <optix_types.h>

#include "optix_aux.h"
#include "optix_dev_ptx.h"

static OptixDeviceContext ocontext = nullptr;
static OptixModule omodule = nullptr;
static OptixProgramGroup oprograms[3] = { nullptr, nullptr, nullptr };
static OptixPipeline opipeline = nullptr;

#define CHECK_OPTIX(A, B) \
    if ( A != OPTIX_SUCCESS ) { \
        std::cerr << "\n[OPTIX ERROR] in " << __FILE__ << "\nLine " << __LINE__ << ": " << B; \
        throw std::runtime_error(B); \
    }

#define CHECK_CUDA(A, B) \
    if ( A != hipSuccess ) { \
        std::cerr << "\n[CUDA ERROR] in " << __FILE__ << "\nLine " << __LINE__ << ": " << B; \
        throw std::runtime_error(B); \
    }

__forceinline__ void __initialize_context() {
    if ( !ocontext ) {
        // std::cout << "Initializing Optix... The first time loading will take some time." << std::endl;
        OptixDeviceContextOptions options = {  };
        // options.validationMode = OPTIX_DEVICE_CONTEXT_VALIDATION_MODE_ALL;
        CHECK_OPTIX( optixInit(), "Initialization Failed." );
        CHECK_OPTIX( optixDeviceContextCreate(nullptr, &options, &ocontext), "Context Creation Failed." );
    }
    OptixModuleCompileOptions moduleCompileOptions = {  };
    OptixPipelineCompileOptions pipelineCompileOptions = {  };
    OptixPipelineLinkOptions pipelineLinkOptions = {  };
    if ( !omodule ) {
        moduleCompileOptions.maxRegisterCount = 128;
        // moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
        // moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_FULL;
        // pipelineCompileOptions.exceptionFlags = OPTIX_EXCEPTION_FLAG_DEBUG;
        
        moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_3;
        moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;
        pipelineCompileOptions.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;

        // optixDeviceContextSetLogCallback(
        //     ocontext,
        //     [](unsigned int level, const char* tag, const char* message, void*) {
        //         printf("[OptiX][%d][%s] %s\n", level, tag, message);
        //     },
        //     nullptr,
        //     4  // Maximum verbosity level
        // );

        pipelineCompileOptions.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
        pipelineCompileOptions.usesMotionBlur = false;
        pipelineCompileOptions.numPayloadValues = 2;
        pipelineCompileOptions.numAttributeValues = 8;
        pipelineCompileOptions.pipelineLaunchParamsVariableName = "optixLaunchParams";

        pipelineLinkOptions.maxTraceDepth = 1;

        char log[2048] = "OptiX Create Module Failed: ";
        size_t sizeof_log = sizeof(log) - strlen(log);

        CHECK_OPTIX( optixModuleCreate(ocontext, &moduleCompileOptions, &pipelineCompileOptions, (const char*)optix_dev_ptx, optix_dev_ptx_len, log + strlen(log), &sizeof_log, &omodule), log );
    }

    if ( !oprograms[0] ) {
        OptixProgramGroupOptions pgOptions = {};
        OptixProgramGroupDesc pgDesc = {};
        pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
        pgDesc.raygen.module = omodule;
        pgDesc.raygen.entryFunctionName = "__raygen__rg";

        char log[2048] = "Optix Create RayGen Program Failed: ";
        size_t sizeof_log = sizeof(log) - strlen(log);
        CHECK_OPTIX( optixProgramGroupCreate(ocontext, &pgDesc, 1, &pgOptions, log, &sizeof_log, &oprograms[0]), log );
    }

    if ( !oprograms[1] ) {
        OptixProgramGroupOptions pgOptions = {};
        OptixProgramGroupDesc pgDesc = {};
        pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
        pgDesc.raygen.module = omodule;
        pgDesc.raygen.entryFunctionName = "__miss__far";

        char log[2048] = "Optix Create Miss Program Failed: ";
        size_t sizeof_log = sizeof(log) - strlen(log);
        CHECK_OPTIX( optixProgramGroupCreate(ocontext, &pgDesc, 1, &pgOptions, log, &sizeof_log, &oprograms[1]), log );
    }

    if ( !oprograms[2] ) {
        OptixProgramGroupOptions pgOptions = {};
        OptixProgramGroupDesc pgDesc = {};
        pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
        pgDesc.hitgroup.moduleCH = nullptr;
        pgDesc.hitgroup.moduleAH = omodule;
        pgDesc.hitgroup.moduleIS = omodule;
        pgDesc.hitgroup.entryFunctionNameAH = "__anyhit__ch";
        pgDesc.hitgroup.entryFunctionNameIS = "__intersection__aabb";

        char log[2048] = "Optix Create Hit Program Failed: ";
        size_t sizeof_log = sizeof(log) - strlen(log);
        CHECK_OPTIX( optixProgramGroupCreate(ocontext, &pgDesc, 1, &pgOptions, log, &sizeof_log, &oprograms[2]), log );
    }

    if ( !opipeline ) {
        char log[2048] = "Optix Create Pipeline Failed: ";
        size_t sizeof_log = sizeof(log) - strlen(log);
        CHECK_OPTIX( optixPipelineCreate(ocontext, &pipelineCompileOptions, &pipelineLinkOptions, oprograms, 3, log, &sizeof_log, &opipeline), log );

        CHECK_OPTIX( optixPipelineSetStackSize(opipeline, 2 * 1024, 2 * 1024, 2 * 1024, 1), "Optix Pipeline Set Stack Size Failed." );
    }
}

std::tuple<unsigned long long, unsigned long long> OptixBuildBVH(
    const torch::Tensor& aabbs
) {
    __initialize_context();
    OptixTraversableHandle out_handle;
    hipDeviceptr_t out_pointer;

    OptixBuildInput buildInput = {};
    buildInput.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;
    
    const hipDeviceptr_t bboxes = reinterpret_cast<const hipDeviceptr_t>(aabbs.contiguous().data_ptr<float>());
    buildInput.customPrimitiveArray.aabbBuffers = &bboxes;
    buildInput.customPrimitiveArray.numPrimitives = aabbs.size(0);
    buildInput.customPrimitiveArray.strideInBytes = sizeof(OptixAabb);

    const unsigned int flags[] = { OPTIX_GEOMETRY_FLAG_REQUIRE_SINGLE_ANYHIT_CALL };
    buildInput.customPrimitiveArray.flags = flags;
    buildInput.customPrimitiveArray.numSbtRecords = 1;

    OptixAccelBuildOptions buildOptions = {};
    buildOptions.buildFlags = OPTIX_BUILD_FLAG_PREFER_FAST_TRACE;
    buildOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

    OptixAccelBufferSizes bufferSizes;
    CHECK_OPTIX( optixAccelComputeMemoryUsage(ocontext,
        &buildOptions,
        &buildInput,
        1,  // num_build_inputs
        &bufferSizes
    ), "Computing Acceleration Structure Memory Usage Failed." );

    hipDeviceptr_t tempBuffer;
    CHECK_CUDA( hipMalloc(&tempBuffer, bufferSizes.tempSizeInBytes), "Temporary Buffer Initialization Failed." );
    CHECK_CUDA( hipMalloc(&out_pointer, bufferSizes.outputSizeInBytes), "BVH Memory Allocation Failed." );
    
    CHECK_OPTIX( optixAccelBuild(ocontext,
        0,
        &buildOptions,
        &buildInput,
        1,
        (hipDeviceptr_t)tempBuffer,
        bufferSizes.tempSizeInBytes,
        (hipDeviceptr_t)out_pointer,
        bufferSizes.outputSizeInBytes,
        &out_handle,
        nullptr, 0
    ), "BVH Acceleration Structure Building Failed.");

    CHECK_CUDA( hipCtxSynchronize(), "Context Synchronization Failed." );
    CHECK_CUDA( hipFree(tempBuffer), "Cleaning Up Temporary Buffer Failed." );

    return std::make_tuple(out_handle, out_pointer);
}

void OptixReleaseBVH(
    unsigned long long handle, 
    unsigned long long pointer
) {
    __initialize_context();

    CHECK_CUDA( hipFree((hipDeviceptr_t)pointer), "Releasing Handle Failed." );
    return;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor> OptixVisibilityEstimatorBackwardCUDA(
    unsigned long long handle, unsigned long long pointer, 
    const torch::Tensor& means3D, 
    const torch::Tensor& geovalues, 
    const torch::Tensor& scales, 
    const torch::Tensor& rots, 
    const torch::Tensor& start_coords, 
    const torch::Tensor& stop_coords, 
    const torch::Tensor& out_visibility, 
    const torch::Tensor& gradient, 
    const float min_decay
) {
    __initialize_context();

    const auto n_rays = start_coords.size(0);
    const auto n_elements = means3D.size(0);
    auto float_opts = means3D.options().dtype(torch::kFloat32);

    auto dL_dmeans3D = torch::full({n_elements, 3}, 0.0f, float_opts);
    auto dL_dgeovalues = torch::full({n_elements, 1}, 0.0f, float_opts);
    auto dL_dscales = torch::full({n_elements, 2}, 0.0f, float_opts);
    auto dL_drots = torch::full({n_elements, 4}, 0.0f, float_opts);
    auto dL_dstart_coords = torch::full({n_rays, 3}, 0.0f, float_opts);
    auto dL_dstop_coords = torch::full({n_rays, 3}, 0.0f, float_opts);

    OptixTraversableHandle ohandle = (OptixTraversableHandle)(handle);

    OptixShaderBindingTable sbt = {};
    constexpr int sbt_record_size = (OPTIX_SBT_RECORD_HEADER_SIZE / OPTIX_SBT_RECORD_ALIGNMENT + 1) * OPTIX_SBT_RECORD_ALIGNMENT;
    char record[sbt_record_size * 3];
    optixSbtRecordPackHeader(oprograms[0], record);
    optixSbtRecordPackHeader(oprograms[1], record + sbt_record_size);
    optixSbtRecordPackHeader(oprograms[2], record + sbt_record_size * 2);

    hipDeviceptr_t dsbt;
    CHECK_CUDA( hipMalloc(&dsbt, sbt_record_size * 3), "Memory Allocation Failed." );
    CHECK_CUDA( hipMemcpyHtoD(dsbt, record, sbt_record_size * 3), "Memory Copy Failed." );
    
    sbt.raygenRecord = dsbt;
    sbt.missRecordBase = dsbt + sbt_record_size;
    sbt.hitgroupRecordBase = dsbt + sbt_record_size * 2;
    sbt.missRecordStrideInBytes = sbt.hitgroupRecordStrideInBytes = sbt_record_size;
    sbt.missRecordCount = sbt.hitgroupRecordCount = 1;

    LaunchParams p = {};
    p.handle = ohandle;
    p.mode = true;
    p.batch_size = n_rays;
    p.min_decay = min_decay;
    p.means3D = (hipDeviceptr_t)(means3D.contiguous().data_ptr<float>());
    p.geovalues = (hipDeviceptr_t)(geovalues.contiguous().data_ptr<float>());
    p.scales = (hipDeviceptr_t)(scales.contiguous().data_ptr<float>());
    p.rots = (hipDeviceptr_t)(rots.contiguous().data_ptr<float>());
    p.start_coords = (hipDeviceptr_t)(start_coords.contiguous().data_ptr<float>());
    p.stop_coords = (hipDeviceptr_t)(stop_coords.contiguous().data_ptr<float>());
    p.out_visibility = (hipDeviceptr_t)(out_visibility.contiguous().data_ptr<float>());

    p.gradient = (hipDeviceptr_t)(gradient.contiguous().data_ptr<float>());
    p.dL_dmeans3D = (hipDeviceptr_t)(dL_dmeans3D.contiguous().data_ptr<float>());
    p.dL_dgeovalues = (hipDeviceptr_t)(dL_dgeovalues.contiguous().data_ptr<float>());
    p.dL_dscales = (hipDeviceptr_t)(dL_dscales.contiguous().data_ptr<float>());
    p.dL_drots = (hipDeviceptr_t)(dL_drots.contiguous().data_ptr<float>());
    p.dL_dstart_coords = (hipDeviceptr_t)(dL_dstart_coords.contiguous().data_ptr<float>());
    p.dL_dstop_coords = (hipDeviceptr_t)(dL_dstop_coords.contiguous().data_ptr<float>());
    
    hipDeviceptr_t dp;
    CHECK_CUDA( hipMalloc(&dp, sizeof(LaunchParams)), "Memory Allocation Failed." );
    CHECK_CUDA( hipMemcpyHtoD(dp, &p, sizeof(LaunchParams)), "Memory Copy Failed." );

    if ( n_rays <= 1024 ) {
        CHECK_OPTIX( optixLaunch(opipeline, 0, (hipDeviceptr_t)dp, sizeof(LaunchParams), &sbt, n_rays, 1, 1), "Optix Launch Failed." );
    } else if ( n_rays <= 1024 * 1024 ) {
        CHECK_OPTIX( optixLaunch(opipeline, 0, (hipDeviceptr_t)dp, sizeof(LaunchParams), &sbt, 1024, (n_rays + 1024 - 1) / 1024, 1), "Optix Launch Failed." );
    } else {
        CHECK_OPTIX( optixLaunch(opipeline, 0, (hipDeviceptr_t)dp, sizeof(LaunchParams), &sbt, 1024, 1024, (n_rays + (1024 * 1024) - 1) / (1024 * 1024)), "Optix Launch Failed." );
    }
    CHECK_CUDA( hipCtxSynchronize(), "Context Synchronization Failed." );
    CHECK_CUDA( hipFree(dp), "Memory Release Failed." );
    CHECK_CUDA( hipFree(dsbt), "Memory Release Failed." );

    return std::make_tuple(dL_dmeans3D, dL_dgeovalues, dL_dscales, dL_drots, dL_dstart_coords, dL_dstop_coords);
}

torch::Tensor OptixVisibilityEstimatorForwardCUDA(
    unsigned long long handle, unsigned long long pointer, 
    const torch::Tensor& means3D, 
    const torch::Tensor& geovalues, 
    const torch::Tensor& scales, 
    const torch::Tensor& rots, 
    const torch::Tensor& start_coords, 
    const torch::Tensor& stop_coords, 
    const float min_decay
) {
    __initialize_context();

    const auto n_rays = start_coords.size(0);
    const auto n_elements = means3D.size(0);
    auto float_opts = means3D.options().dtype(torch::kFloat32);
    auto out_visibility = torch::full({n_rays}, 0.0f, float_opts);

    OptixTraversableHandle ohandle = (OptixTraversableHandle)(handle);

    OptixShaderBindingTable sbt = {};
    constexpr int sbt_record_size = (OPTIX_SBT_RECORD_HEADER_SIZE / OPTIX_SBT_RECORD_ALIGNMENT + 1) * OPTIX_SBT_RECORD_ALIGNMENT;
    char record[sbt_record_size * 3];
    optixSbtRecordPackHeader(oprograms[0], record);
    optixSbtRecordPackHeader(oprograms[1], record + sbt_record_size);
    optixSbtRecordPackHeader(oprograms[2], record + sbt_record_size * 2);

    hipDeviceptr_t dsbt;
    CHECK_CUDA( hipMalloc(&dsbt, sbt_record_size * 3), "Memory Allocation Failed." );
    CHECK_CUDA( hipMemcpyHtoD(dsbt, record, sbt_record_size * 3), "Memory Copy Failed." );
    
    sbt.raygenRecord = dsbt;
    sbt.missRecordBase = dsbt + sbt_record_size;
    sbt.hitgroupRecordBase = dsbt + sbt_record_size * 2;
    sbt.missRecordStrideInBytes = sbt.hitgroupRecordStrideInBytes = sbt_record_size;
    sbt.missRecordCount = sbt.hitgroupRecordCount = 1;

    LaunchParams p = {};
    p.handle = ohandle;
    p.mode = false;
    p.batch_size = n_rays;
    p.means3D = (hipDeviceptr_t)(means3D.contiguous().data_ptr<float>());
    p.geovalues = (hipDeviceptr_t)(geovalues.contiguous().data_ptr<float>());
    p.scales = (hipDeviceptr_t)(scales.contiguous().data_ptr<float>());
    p.rots = (hipDeviceptr_t)(rots.contiguous().data_ptr<float>());
    p.start_coords = (hipDeviceptr_t)(start_coords.contiguous().data_ptr<float>());
    p.stop_coords = (hipDeviceptr_t)(stop_coords.contiguous().data_ptr<float>());
    p.min_decay = min_decay;
    p.out_visibility = (hipDeviceptr_t)(out_visibility.contiguous().data_ptr<float>());
    
    hipDeviceptr_t dp;
    CHECK_CUDA( hipMalloc(&dp, sizeof(LaunchParams)), "Memory Allocation Failed." );
    CHECK_CUDA( hipMemcpyHtoD(dp, &p, sizeof(LaunchParams)), "Memory Copy Failed." );

    if ( n_rays <= 1024 ) {
        CHECK_OPTIX( optixLaunch(opipeline, 0, (hipDeviceptr_t)dp, sizeof(LaunchParams), &sbt, n_rays, 1, 1), "Optix Launch Failed." );
    } else if ( n_rays <= 1024 * 1024 ) {
        CHECK_OPTIX( optixLaunch(opipeline, 0, (hipDeviceptr_t)dp, sizeof(LaunchParams), &sbt, 1024, (n_rays + 1024 - 1) / 1024, 1), "Optix Launch Failed." );
    } else {
        CHECK_OPTIX( optixLaunch(opipeline, 0, (hipDeviceptr_t)dp, sizeof(LaunchParams), &sbt, 1024, 1024, (n_rays + (1024 * 1024) - 1) / (1024 * 1024)), "Optix Launch Failed." );
    }
    CHECK_CUDA( hipCtxSynchronize(), "Context Synchronization Failed." );
    CHECK_CUDA( hipFree(dp), "Memory Release Failed." );
    CHECK_CUDA( hipFree(dsbt), "Memory Release Failed." );

    return out_visibility;
}